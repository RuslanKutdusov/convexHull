#include "hip/hip_runtime.h"
#include "gpu.hpp"

#include <stdio.h>
#include <vector>

#define CUDA_CHECK_RETURN( value ) {											\
	hipError_t _m_cudaStat = value;										\
	if ( _m_cudaStat != hipSuccess ) {										\
		fprintf( stderr, "Error '%s' at line %d in file %s\n",					\
				hipGetErrorString( _m_cudaStat ), __LINE__, __FILE__ );		\
		exit( 1 );															\
	} }

namespace gpu
{


//
const int MAX_GPU_COUNT = 8;

// TODO: cuda arrays??
texture< FP, 1, hipReadModeElementType > g_texturePoints;
texture< FP, 1, hipReadModeElementType > g_textureVals;
texture< FP, 1, hipReadModeElementType > g_textureHyperplanes;


//
__global__ void kernel1( FP* hyperplanes, FP* points, FP* vals, const size_t n, size_t numberOfHyperplanes, size_t numberOfPoints )
{
	size_t hyperplaneIndex = blockIdx.x * blockDim.x + threadIdx.x;

	if( hyperplaneIndex >= numberOfHyperplanes )
		return;

	size_t dimX = n - 1;
	// now its offset to hyperplane in 'hyperplanes' array, to remove redundant multiplication at every string
	size_t offsetToHyperplane = hyperplaneIndex * ( n + 1 );

	FP resultDistance = 0.0;
	
	for( size_t k = 0; k < numberOfPoints; k++ )
	{
		FP d = 0.0;

		// dot product of point and normal is distance
		for( size_t j = 0; j < dimX; j++ ) 
			d += points[ k * dimX + j ] * hyperplanes[ offsetToHyperplane + j ]; // TODO: shared memory in k loop?
		d += vals[ k ] * hyperplanes[ offsetToHyperplane + n - 1 ]; 

		if( d > resultDistance )
			resultDistance = d;
	}

	hyperplanes[ offsetToHyperplane + n ] = resultDistance;
}


// TODO: pair?
__global__ void kernel1_1( FP** hyperplanes, int deviceCount, size_t n, size_t numberOfHyperplanes )
{
	size_t hyperplaneIndex = blockIdx.x * blockDim.x + threadIdx.x;

	if( hyperplaneIndex >= numberOfHyperplanes )
		return;

	size_t offset = hyperplaneIndex * ( n + 1 ) + n;

	FP resultDistance = hyperplanes[ 0 ][ offset ];
	for( int i = 1; i < deviceCount; i++ )
	{
		if( hyperplanes[ i ][ offset ] > resultDistance )
			resultDistance = hyperplanes[ i ][ offset ];
	}
	hyperplanes[ 0 ][ offset ] = resultDistance;
}


// sending dimX as argument is to reduce registers usage
__global__ void kernel2( FP* hyperplanes, FP* points, FP* vals, size_t n, size_t dimX, size_t numberOfHyperplanes, size_t numberOfPoints )
{
	size_t k = blockIdx.x * blockDim.x + gridDim.x * blockDim.x * blockIdx.y + threadIdx.x;
	if( k >= numberOfPoints )
		return;

	FP funcVal = vals[ k ];
	FP convexVal = funcVal;

	for( size_t i = 0; i < numberOfHyperplanes; i++ )
	{
		FP val = 0.0;
		size_t offsetToHyperplane = i * ( n + 1 );
		// xi - iter->first
		// Ni - hyperplane normal
		// val = x(n - 1) = ( -N0*x0 - N1*x1 - ... - N(n - 2)*x(n - 2) + xn ) / N(n - 1)
		for( size_t j = 0; j < dimX; j++ )
			val -= points[ k * dimX + j ] * hyperplanes[ offsetToHyperplane + j ];
		val += hyperplanes[ offsetToHyperplane + n ];
		val /= hyperplanes[ offsetToHyperplane + n - 1 ] + EPSILON;

		if( i == 0 )
		{
			convexVal = val;
			continue;
		}

		if( val < convexVal && val >= funcVal )
			convexVal = val;
	}

	vals[ k ] = convexVal;
}


//
void getGridAndBlockDim( int n, dim3& gridDim, dim3& blockDim, int device )
{
	// gpu hardware limits for compute caps 1.2 and 2.0
	const int warpSize = 32;
	const int blocksPerSM = 8;

	hipDeviceProp_t deviceProp;
	CUDA_CHECK_RETURN( hipGetDeviceProperties( &deviceProp, device ) );

	int warpCount = ( n / warpSize ) + ( ( ( n % warpSize ) == 0 ) ? 0 : 1 );

	int threadsPerBlock = deviceProp.maxThreadsPerMultiProcessor / blocksPerSM;
	int warpsPerBlock = threadsPerBlock / warpSize;

	int blockCount = ( warpCount / warpsPerBlock ) + ( ( ( warpCount % warpsPerBlock ) == 0 ) ? 0 : 1 );

	blockDim = dim3( threadsPerBlock, 1, 1 );

	gridDim = dim3( blockCount, 1, 1 );

	if( blockCount > deviceProp.maxGridSize[ 0 ] )
	{
		gridDim.x = gridDim.y = sqrt( blockCount );
		if( gridDim.x * gridDim.x < blockCount )
			gridDim.x += 1;
	}

	printf( "GPU%d: %s, Task size: %d, warp number: %d, threads per block: %d, warps per block: %d, grid: (%d, %d, 1)\n", device, deviceProp.name, n, warpCount, threadsPerBlock, warpsPerBlock, gridDim.x, gridDim.y );
}


//
__host__ void makeConvex( ScalarFunction& func, const size_t& dimX, const size_t& numberOfPoints )
{
	if( dimX == 0 )
		return;
	
	FP dFi = PI / ( numberOfPoints - 1 );

	size_t n = dimX + 1; // space dimension

	size_t numberOfHyperplanes = pow( numberOfPoints, n - 1 );

	// first x0.. x(n - 2) elements are independent vars. in 2D it will be x
	// x(n - 1) element dependent var. . in 2D it will be y
	// xn - constant, represents distance between O and hyperplane
	size_t hyperplanesArraySize = numberOfHyperplanes * ( n + 1 );
	size_t hyperplanesArrayLength = hyperplanesArraySize * sizeof( FP );
	FP* hyperplanes = new FP[ hyperplanesArraySize ];

	size_t pointsArraySize = dimX * func.size();
	FP* points = new FP[ pointsArraySize ];

	size_t valsArraySize = func.size();
	FP* vals = new FP[ valsArraySize ];

	{
		size_t i = 0;
		for( ScalarFunction::iterator iter = func.begin(); iter != func.end(); ++iter, i++ )
		{
			for( size_t j = 0; j < dimX; j++ )
				points[ i * dimX + j ] = iter->first[ j ];

			vals[ i ] = iter->second;
		}
	}

	FPVector fi( dimX, 0.0 );

	for( size_t i = 0; i < numberOfHyperplanes; i++ )
	{
		for( size_t j = 0; j < n; j++ )
		{
			FP* normal = &hyperplanes[ i * ( n + 1 ) ];

			normal[ j ] = 1.0;
			for( size_t k = 0; k < j; k++ )
				normal[ j ] *= sin( fi[ k ] );

			if( j != n - 1 )
				normal[ j ] *= cos( fi[ j ] );
		}

		// not good enough
		bool shift = true;
		for( size_t k = 0; ( k < dimX ) && shift; k++ )
		{
			if( fabs( fi[ k ] - PI ) <= EPSILON )
			{
				fi[ k ] = 0.0;
				shift = true;	
			}
			else
			{
				fi[ k ] += dFi;
				shift = false;
			}

			if( fi[ k ] - PI > EPSILON )
				fi[ k ] = PI;
		}
	}

	//
	int deviceCount = 0;
	CUDA_CHECK_RETURN( hipGetDeviceCount( &deviceCount ) );
	printf( "Available device count: %d\n", deviceCount );
	if( deviceCount > MAX_GPU_COUNT )
	{
		printf( "Too much GPUs %d\n", deviceCount );
		deviceCount = MAX_GPU_COUNT;
	}

	// особенность суперкомпьютера Уран, 8 видеокарт одного узла по сути разбиты на 2 части
	// такие, что для видеокарт одной части возможен peer access, но для видеокарт из разных частей - нет.
	// определяем для каких видеокарт возможен peer access( они разбиваются на части(группы) ). 
	// выбираем ту группу для работы, кол-во видеокарт в которой больше, чем в другой.
	std::vector< int > devicesGroups[ 2 ];

	// 
	{
		CUDA_CHECK_RETURN( hipSetDevice( 0 ) );
		devicesGroups[ 0 ].push_back( 0 );
		for( int j = 1; j < deviceCount; j++ )
		{
			int accessible;
			hipDeviceCanAccessPeer( &accessible, j, 0 );
			if( accessible )
				devicesGroups[ 0 ].push_back( j );
			else
				devicesGroups[ 1 ].push_back( j );
		}
	}

	std::vector< int >& usedDevices = ( devicesGroups[ 0 ].size() > devicesGroups[ 1 ].size() ) ? devicesGroups[ 0 ] : devicesGroups[ 1 ];

	deviceCount = usedDevices.size();

	// enabling peer access
	CUDA_CHECK_RETURN( hipSetDevice( usedDevices[ 0 ] ) );
	for( int i = 1; i < ( int )usedDevices.size(); i++ )	
		CUDA_CHECK_RETURN( hipDeviceEnablePeerAccess( usedDevices[ i ], 0 ) );

	printf( "Used device count: %d\n", deviceCount );

	const size_t pointsPerDevice = func.size() / deviceCount;
	FP* d_hyperplanes[ MAX_GPU_COUNT ];
	FP* d_points[ MAX_GPU_COUNT ];
	FP* d_vals[ MAX_GPU_COUNT ];
	dim3 gridDim, blockDim;

	//
	printf( "Running first kernel...\n" );
	for( int i = 0; i < deviceCount; i++ )
	{
		int device = usedDevices[ i ];
		CUDA_CHECK_RETURN( hipSetDevice( device ) );

		//
		CUDA_CHECK_RETURN( hipMalloc( &d_hyperplanes[ i ], hyperplanesArrayLength ) );
		//CUDA_CHECK_RETURN( hipBindTexture( NULL, g_textureHyperplanes[ i ], d_hyperplanes[ i ], hyperplanesArrayLength ) );

		if( i == 0 )
		{
			CUDA_CHECK_RETURN( hipMemcpy( d_hyperplanes[ i ], hyperplanes, hyperplanesArrayLength, hipMemcpyHostToDevice ) );
		}
		else
		{
			// TODO: smart copying, pair
			int lastDevice = usedDevices[ i - 1 ];
			CUDA_CHECK_RETURN( hipMemcpyPeer( d_hyperplanes[ i ], device, d_hyperplanes[ i - 1 ], lastDevice, hyperplanesArrayLength ) );
		}

		int arrayOffset = pointsPerDevice * i;
		int pointsPerCurrentDevice = pointsPerDevice;
		if( i == deviceCount - 1 )
			pointsPerCurrentDevice = func.size() - pointsPerDevice * i;

		//
		int bytesCount = pointsPerCurrentDevice * dimX * sizeof( FP );
		CUDA_CHECK_RETURN( hipMalloc( &d_points[ i ], bytesCount ) );
		CUDA_CHECK_RETURN( hipMemcpy( d_points[ i ], points + arrayOffset * dimX, bytesCount, hipMemcpyHostToDevice ) );
		//CUDA_CHECK_RETURN( hipBindTexture( NULL, g_texturePoints[ i ], d_points[ i ], bytesCount ) );

		//
		bytesCount = pointsPerCurrentDevice * sizeof( FP );
		CUDA_CHECK_RETURN( hipMalloc( &d_vals[ i ], bytesCount ) );
		CUDA_CHECK_RETURN( hipMemcpy( d_vals[ i ], vals + arrayOffset, bytesCount, hipMemcpyHostToDevice ) );
		//CUDA_CHECK_RETURN( hipBindTexture( NULL, g_textureVals[ i ], d_vals, bytesCount ) );

		// run first kernel
		getGridAndBlockDim( numberOfHyperplanes, gridDim, blockDim, device );
		kernel1<<< gridDim, blockDim >>>( d_hyperplanes[ i ], d_points[ i ], d_vals[ i ], n, numberOfHyperplanes, pointsPerCurrentDevice );
	}

	//
	printf( "Synchronizing...\n" );
	for( int i = 0; i < deviceCount; i++ )
	{
		int device = usedDevices[ i ];
		CUDA_CHECK_RETURN( hipSetDevice( device ) );
		CUDA_CHECK_RETURN( hipDeviceSynchronize() );
		CUDA_CHECK_RETURN( hipGetLastError() );
	}

	//
	{
		//
		printf( "Running second kernel...\n" );

		int device = usedDevices[ 0 ];
		CUDA_CHECK_RETURN( hipSetDevice( device ) );

		FP** hostAllocatedMem;
		hipHostAlloc( ( void** )&hostAllocatedMem, deviceCount * sizeof( FP* ), hipHostMallocDefault );
		for( int i = 0; i < deviceCount; i++ )
			hostAllocatedMem[ i ] = d_hyperplanes[ i ];

		getGridAndBlockDim( numberOfHyperplanes, gridDim, blockDim, device );
		kernel1_1<<< gridDim, blockDim >>>( hostAllocatedMem, deviceCount, n, numberOfHyperplanes );

		printf( "Synchronizing...\n" );
		CUDA_CHECK_RETURN( hipDeviceSynchronize() );
		CUDA_CHECK_RETURN( hipGetLastError() );
		CUDA_CHECK_RETURN( hipHostFree( hostAllocatedMem ) );
	}

	//
	printf( "Running third kernel...\n" );
	for( int i = 0; i < deviceCount; i++ )
	{
		int device = usedDevices[ i ];
		CUDA_CHECK_RETURN( hipSetDevice( device ) );

		// copy hyperplanes from first device to others
		if( i != 0 )
		{
			// TODO: smart copying, pair
			int lastDevice = usedDevices[ i - 1 ];
			CUDA_CHECK_RETURN( hipMemcpyPeer( d_hyperplanes[ i ], device, d_hyperplanes[ i - 1 ], lastDevice, hyperplanesArrayLength ) );
		}

		int pointsPerCurrentDevice = pointsPerDevice;
		if( i == deviceCount - 1 )
			pointsPerCurrentDevice = func.size() - pointsPerDevice * i;

		getGridAndBlockDim( pointsPerCurrentDevice, gridDim, blockDim, device );
		kernel2<<< gridDim, blockDim >>>( d_hyperplanes[ i ], d_points[ i ], d_vals[ i ], n, dimX, numberOfHyperplanes, pointsPerCurrentDevice );
	}

	//
	printf( "Synchronizing...\n" );
	for( int i = 0; i < deviceCount; i++ )
	{
		int device = usedDevices[ i ];
		CUDA_CHECK_RETURN( hipSetDevice( device ) );
		CUDA_CHECK_RETURN( hipDeviceSynchronize() );
		CUDA_CHECK_RETURN( hipGetLastError() );

		int arrayOffset = pointsPerDevice * i;
		int pointsPerCurrentDevice = pointsPerDevice;
		if( i == deviceCount - 1 )
			pointsPerCurrentDevice = func.size() - pointsPerDevice * i;

		//
		int bytesCount = pointsPerCurrentDevice * sizeof( FP );
		printf( "Copying result from GPU%d, %d bytes\n", device, bytesCount );
		CUDA_CHECK_RETURN( hipMemcpy( vals + arrayOffset, d_vals[ i ], bytesCount, hipMemcpyDeviceToHost ) );
	}

	// int device = usedDevices[ 0 ];
	// CUDA_CHECK_RETURN( hipSetDevice( device ) );

	// FP* d_points_;
	// CUDA_CHECK_RETURN( hipMalloc( &d_points_, pointsArrayLength ) );
	// CUDA_CHECK_RETURN( hipMemcpy( d_points_, points, pointsArrayLength, hipMemcpyHostToDevice ) );

	// FP* d_vals_;
	// CUDA_CHECK_RETURN( hipMalloc( &d_vals_, valsArrayLength ) );
	// CUDA_CHECK_RETURN( hipMemcpy( d_vals_, vals, valsArrayLength, hipMemcpyHostToDevice ) );

	// // run second kernel
	// getGridAndBlockDim( func.size(), gridDim, blockDim, device );
	// kernel2<<< gridDim, blockDim >>>( d_hyperplanes[ 0 ], d_points_, d_vals_, n, dimX, numberOfHyperplanes, func.size() );

	// CUDA_CHECK_RETURN( hipDeviceSynchronize() );
	// CUDA_CHECK_RETURN( hipGetLastError() );

	// CUDA_CHECK_RETURN( hipMemcpy( vals, d_vals_, valsArraySize * sizeof( FP ), hipMemcpyDeviceToHost ) );
	// CUDA_CHECK_RETURN( hipGetLastError() );


	// ???
	//func.clear();

	printf( "Storing result...\n" );	
	for( size_t k = 0; k < func.size(); k++ )
	{
		FPVector x( &points[ k * dimX ], &points[ ( k + 1 ) * dimX ] );
		func.define( x ) = vals[ k ];
	}

	for( int i = 0; i < deviceCount; i++ )
	{
		int device = usedDevices[ i ];
		CUDA_CHECK_RETURN( hipSetDevice( device ) ); 
		CUDA_CHECK_RETURN( hipFree( ( void* )d_hyperplanes[ i ] ) );
		CUDA_CHECK_RETURN( hipFree( ( void* )d_points[ i ] ) );
		CUDA_CHECK_RETURN( hipFree( ( void* )d_vals[ i ] ) );

		//
		CUDA_CHECK_RETURN( hipDeviceReset() );
		CUDA_CHECK_RETURN( hipGetLastError() );
	}

	delete[] hyperplanes;
	delete[] points;
	delete[] vals;

	printf( "Done\n" );
}

}