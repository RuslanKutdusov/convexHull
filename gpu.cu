#include "hip/hip_runtime.h"
#include "gpu.hpp"

#include <stdio.h>

#define CUDA_CHECK_RETURN( value ) {											\
	hipError_t _m_cudaStat = value;										\
	if ( _m_cudaStat != hipSuccess ) {										\
		fprintf( stderr, "Error '%s' at line %d in file %s\n",					\
				hipGetErrorString( _m_cudaStat ), __LINE__, __FILE__ );		\
		exit( 1 );															\
	} }

texture< FP, 1, hipReadModeElementType > g_texturePoints;
texture< FP, 1, hipReadModeElementType > g_textureVals;
texture< FP, 1, hipReadModeElementType > g_textureHyperplanes;

__global__ void kernel1( FP* hyperplanes, const size_t n, size_t numberOfHyperplanes, size_t numberOfPoints )
{
	size_t hyperplaneIndex = blockIdx.x * blockDim.x + threadIdx.x;

	if( hyperplaneIndex >= numberOfHyperplanes )
		return;

	size_t dimX = n - 1;
	// now its offset to hyperplane in 'hyperplanes' array, to remove redundant multiplication at every string
	size_t offsetToHyperplane = hyperplaneIndex * ( n + 1 );

	FP resultDistance = 0.0;
	
	for( size_t k = 0; k < numberOfPoints; k++ )
	{
		FP d = 0.0;

		// dot product of point and normal is distance
		for( size_t j = 0; j < dimX; j++ )
			d += tex1Dfetch( g_texturePoints, k * dimX + j ) * hyperplanes[ offsetToHyperplane + j ]; 
		d += tex1Dfetch( g_textureVals, k ) * hyperplanes[ offsetToHyperplane + n - 1 ]; 

		if( d > resultDistance )
			resultDistance = d;
	}

	hyperplanes[ offsetToHyperplane + n ] = resultDistance;
}


//
__global__ void kernel2( FP* vals, size_t n, size_t numberOfHyperplanes, size_t numberOfPoints, size_t taskSize )
{
	size_t pointIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if( pointIndex >= numberOfPoints )
		return;

	size_t dimX = n - 1;	

	for( size_t k = pointIndex; k < pointIndex + taskSize; k++ )
	{
		FP funcVal = vals[ k ];
		FP convexVal = funcVal;

		for( size_t i = 0; i < numberOfHyperplanes; i++ )
		{
			FP val = 0.0;
			size_t offsetToHyperplane = i * ( n + 1 );
			// xi - iter->first
			// Ni - hyperplane normal
			// val = x(n - 1) = ( -N0*x0 - N1*x1 - ... - N(n - 2)*x(n - 2) + xn ) / N(n - 1)
			for( size_t j = 0; j < dimX; j++ )
				val -= tex1Dfetch( g_texturePoints, k * dimX + j ) * tex1Dfetch( g_textureHyperplanes, offsetToHyperplane + j );
			val += tex1Dfetch( g_textureHyperplanes, offsetToHyperplane + n );
			val /= tex1Dfetch( g_textureHyperplanes, offsetToHyperplane + n - 1 ) + EPSILON;

			if( i == 0 )
			{
				convexVal = val;
				continue;
			}

			if( val < convexVal && val >= funcVal )
				convexVal = val;
		}

		vals[ k ] = convexVal;
	}
}

__host__ void makeConvexGPU_( ScalarFunction& func, const size_t& dimX, const size_t& numberOfPoints )
{
	if( dimX == 0 )
		return;
	
	FP dFi = PI / ( numberOfPoints - 1 );

	size_t n = dimX + 1; // space dimension

	size_t numberOfHyperplanes = pow( numberOfPoints, n - 1 );

	// first x0.. x(n - 2) elements are independent vars. in 2D it will be x
	// x(n - 1) element dependent var. . in 2D it will be y
	// xn - constant, represents distance between O and hyperplane
	size_t hyperplanesSize = numberOfHyperplanes * ( n + 1 );
	FP* hyperplanes = new FP[ hyperplanesSize ];

	size_t pointsSize = dimX * func.size();
	FP* points = new FP[ pointsSize ];

	size_t valsSize = func.size();
	FP* vals = new FP[ valsSize ];

	{
		size_t i = 0;
		for( ScalarFunction::iterator iter = func.begin(); iter != func.end(); ++iter, i++ )
		{
			for( size_t j = 0; j < dimX; j++ )
				points[ i * dimX + j ] = iter->first[ j ];

			vals[ i ] = iter->second;
		}
	}

	FPVector fi( dimX, 0.0 );

	for( size_t i = 0; i < numberOfHyperplanes; i++ )
	{
		for( size_t j = 0; j < n; j++ )
		{
			FP* normal = &hyperplanes[ i * ( n + 1 ) ];

			normal[ j ] = 1.0;
			for( size_t k = 0; k < j; k++ )
				normal[ j ] *= sin( fi[ k ] );

			if( j != n - 1 )
				normal[ j ] *= cos( fi[ j ] );
		}

		// not good enough
		bool shift = true;
		for( size_t k = 0; ( k < dimX ) && shift; k++ )
		{
			if( fabs( fi[ k ] - PI ) <= EPSILON )
			{
				fi[ k ] = 0.0;
				shift = true;	
			}
			else
			{
				fi[ k ] += dFi;
				shift = false;
			}

			if( fi[ k ] - PI > EPSILON )
				fi[ k ] = PI;
		}
	}

	FP* d_hyperplanes;
	CUDA_CHECK_RETURN( hipMalloc( &d_hyperplanes, hyperplanesSize * sizeof( FP ) ) );
	CUDA_CHECK_RETURN( hipMemcpy( d_hyperplanes, hyperplanes, hyperplanesSize * sizeof( FP ), hipMemcpyHostToDevice ) );
	CUDA_CHECK_RETURN( hipBindTexture( NULL, g_textureHyperplanes, d_hyperplanes, hyperplanesSize * sizeof( FP ) ) );

	FP* d_points;
	CUDA_CHECK_RETURN( hipMalloc( &d_points, pointsSize * sizeof( FP ) ) );
	CUDA_CHECK_RETURN( hipMemcpy( d_points, points, pointsSize * sizeof( FP ), hipMemcpyHostToDevice ) );
	CUDA_CHECK_RETURN( hipBindTexture( NULL, g_texturePoints, d_points, pointsSize * sizeof( FP ) ) );

	FP* d_vals;
	CUDA_CHECK_RETURN( hipMalloc( &d_vals, valsSize * sizeof( FP ) ) );
	CUDA_CHECK_RETURN( hipMemcpy( d_vals, vals, valsSize * sizeof( FP ), hipMemcpyHostToDevice ) );
	CUDA_CHECK_RETURN( hipBindTexture( NULL, g_textureVals, d_vals, valsSize * sizeof( FP ) ) );

	const size_t warpSize = 512;

	// run first kernel
	printf( "run kernel1 %u\n", numberOfHyperplanes );
	size_t gridSize = numberOfHyperplanes / warpSize + 1; 
	size_t blockSize = warpSize;
	kernel1<<< gridSize, blockSize >>>( d_hyperplanes, n, numberOfHyperplanes, func.size() );

	CUDA_CHECK_RETURN( hipDeviceSynchronize() );
	CUDA_CHECK_RETURN( hipGetLastError() );
	printf( "fin kernel1\n");


	// run second kernel
	printf( "run kernel2 %u\n", func.size() );
	gridSize = func.size() / warpSize + 1; 
	blockSize = warpSize;
	kernel2<<< gridSize, blockSize >>>( d_vals, n, numberOfHyperplanes, func.size(), 1 );

	CUDA_CHECK_RETURN( hipDeviceSynchronize() );
	CUDA_CHECK_RETURN( hipGetLastError() );
	printf( "fin kernel2\n");

	CUDA_CHECK_RETURN( hipMemcpy( vals, d_vals, valsSize * sizeof( FP ), hipMemcpyDeviceToHost ) );
	CUDA_CHECK_RETURN( hipGetLastError() );

	for( size_t k = 0; k < func.size(); k++ )
	{
		FPVector x( &(points[ k * dimX ]), &(points[ ( k + 1 ) * dimX ]) );
		func.define( x ) = vals[ k ];
	}

	FILE* file = fopen( "data4", "w" );

	for( ScalarFunction::const_iterator iter = func.begin(); iter != func.end(); ++iter )
	{
		fprintf( file, "%g %g %g\n", iter->first[ 0 ], iter->first[ 1 ], iter->second );	
	}

	fclose( file );

	CUDA_CHECK_RETURN( hipFree( ( void* )d_hyperplanes ) );
	CUDA_CHECK_RETURN( hipFree( ( void* )d_points ) );
	CUDA_CHECK_RETURN( hipFree( ( void* )d_vals ) );

	CUDA_CHECK_RETURN( hipDeviceReset() );
	CUDA_CHECK_RETURN( hipGetLastError() );
}